#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define N 10000  

void init_vector(float* vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX * 2.0f - 1.0f; 
    }
}

void softmax_cpu(float* input, float* output, int length) {
    float sum = 0.0f;
    for (int i = 0; i < length; i++) {
        output[i] = expf(input[i]);
        sum += output[i];
    }
    for (int i = 0; i < length; i++) {
        output[i] /= sum;
    }
}

int main() {
    float *h_in, *h_out_cpu, *h_out_gpu;
    float *d_in, *d_out;

    size_t size = N * sizeof(float);
    
    h_in = (float*)malloc(size);
    h_out_cpu = (float*)malloc(size);
    h_out_gpu = (float*)malloc(size);

    srand(time(NULL));
    init_vector(h_in, N);

    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    // cuDNN setup
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t tensorDesc;
    hipdnnCreateTensorDescriptor(&tensorDesc);
    hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, N);

    float alpha = 1.0f, beta = 0.0f;
    hipdnnSoftmaxForward(cudnn, HIPDNN_SOFTMAX_ACCURATE, HIPDNN_SOFTMAX_MODE_INSTANCE, 
                        &alpha, tensorDesc, d_in, &beta, tensorDesc, d_out);

    hipMemcpy(h_out_gpu, d_out, size, hipMemcpyDeviceToHost);

    // Compute softmax on CPU for validation
    softmax_cpu(h_in, h_out_cpu, N);

    // Validate the results
    bool match = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_out_cpu[i] - h_out_gpu[i]) > 1e-5) { 
            match = false;
            break;
        }
    }

    if (match) {
        printf("CPU and GPU softmax outputs match.\n");
    } else {
        printf("CPU and GPU softmax outputs do not match.\n");
    }

    // Cleanup
    hipdnnDestroyTensorDescriptor(tensorDesc);
    hipdnnDestroy(cudnn);

    free(h_in);
    free(h_out_cpu);
    free(h_out_gpu);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}