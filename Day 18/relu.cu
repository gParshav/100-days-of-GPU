#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M 256
#define N 128
#define BLOCK_SIZE 64

void init_matrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        // Generate values in range [-1.0, 1.0]
        mat[i] = 2.0f * ((float)rand() / RAND_MAX) - 1.0f;
    }
}


// CPU implementation of softmax
void relu_cpu(float* input, float* output, int m, int n) {
    
    for(int i=0;i<m;i++){
        for (int j = 0; j < n; j++) {
            int idx = i*n+j;
            output[idx] = max(0.0f, input[idx]);
        }
    }
    
}

// Naïve CUDA softmax kernel
__global__ void relu_gpu(float* input, float* output, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= m) return;

    for (int j = 0; j < n; j++) {
        int idx = row*n+j;
        output[idx] = max(0.0f, input[idx]);
    }
}



int main() {

    float *h_A, *h_C_cpu, *h_C_gpu; //These will be stored on the CPU
    float *d_A, *d_C; //These will be stored on the GPU

    int size_A = M*N*sizeof(float);
    int size_C = M*N*sizeof(float);

    h_A = (float*)malloc(size_A);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    srand(time(NULL));
    init_matrix(h_A, M, N);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, 1); //These are number of threads in x and y inside the block
    dim3 gridDim((M+BLOCK_SIZE-1)/BLOCK_SIZE, 1);


    relu_gpu<<<gridDim, blockDim>>>(d_A, d_C, M, N);
    hipDeviceSynchronize();
    hipMemcpy(h_C_gpu, d_C, size_C, hipMemcpyDeviceToHost);

    // Compute softmax on CPU for validation
    relu_cpu(h_A, h_C_cpu, M, N);

    // Validate results
    bool correct = true;
    for (int i = 0; i < N; i++) {
        
        if (fabs(h_C_cpu[i] - h_C_gpu[i]) > 1e-4) {
            correct = false;
            break;
        }
    }
    
    printf("Results are %s\n", correct ? "correct" : "incorrect");

    // Free memory
    free(h_A);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_C);

    return 0;

}