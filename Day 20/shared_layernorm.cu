#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M 1024
#define N 1024
#define BLOCK_SIZE 256
#define EPSILON 1e-6

void layernorm_cpu(float* A, float* C, int m, int n) {
    for (int i = 0; i < m; i++) {
        float sum = 0.0f;
        for (int j = 0; j < n; j++) {
            sum += A[i * n + j];
        }
        float mean = sum / n;
        float diff_sum = 0.0f;
        for (int j = 0; j < n; j++) {
            diff_sum += (A[i * n + j] - mean) * (A[i * n + j] - mean);
        }
        float var = diff_sum / n;
        float stddev = sqrt(var);
        for (int j = 0; j < n; j++) {
            C[i * n + j] = (A[i * n + j] - mean) / stddev;
        }
    }
}

__global__ void sharedlayernorm_gpu(float* A, float* C, int m, int n) {

    __shared__ float smem[1024];
    int row = blockIdx.x;
    int tidx = threadIdx.x;

    if(row>=m){
        return;
    }


    float lmean = 0.0f;
    float lvar = 0.0f;

    for(int i = tidx; i < n; i+=blockDim.x){
        float curr = A[row*n+i]; 
        lmean += curr;
        lvar += (curr*curr);
    }

    __syncthreads();
    smem[tidx] = lmean;
    __syncthreads();


    for(int stride = blockDim.x/2; stride >0; stride /=2){
        if(tidx < stride){
            smem[tidx] += smem[tidx + stride]; 
        }
        __syncthreads();
    }

    float gmean = smem[0] / n;
    __syncthreads();

    smem[tidx] = lvar;
    __syncthreads();

    for(int stride = blockDim.x; stride > 0; stride /= 2){
        if(tidx < stride){
            smem[tidx] += smem[tidx + stride];

        }
        __syncthreads();
    }

    float gvar = (smem[0]/n) - (gmean * gmean);
    float stddev = rsqrtf(gvar + EPSILON); 
    __syncthreads();


    for(int i = tidx; i < n; i += blockDim.x){
        C[row*n+i] = (A[row*n+i] - gmean) * stddev;
    }
}

void init_matrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main() {
    float *h_A, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_C;

    int size_A = M * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    h_A = (float*)malloc(size_A);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    srand(time(NULL));
    init_matrix(h_A, M, N);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_C, size_C);
    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, 1);
    dim3 gridDim(M, 1);

    double start_cpu = get_time();
    layernorm_cpu(h_A, h_C_cpu, M, N);
    double end_cpu = get_time();
    printf("CPU Time: %f seconds\n", end_cpu - start_cpu);

    double start_gpu = get_time();
    sharedlayernorm_gpu<<<gridDim, blockDim>>>(d_A, d_C, M, N);
    hipDeviceSynchronize();
    double end_gpu = get_time();
    printf("GPU Time: %f seconds\n", end_gpu - start_gpu);

    hipMemcpy(h_C_gpu, d_C, size_C, hipMemcpyDeviceToHost);

    bool correct = true;
    for (int i = 0; i < M * N; i++) {
        if (fabs(h_C_cpu[i] - h_C_gpu[i]) > 1e-4) {
            correct = false;
            break;
        }
    }
    printf("Results are %s\n", correct ? "correct" : "incorrect");

    free(h_A);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_C);

    return 0;
}
