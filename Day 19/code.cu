#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M 256
#define N 128
#define BLOCK_SIZE 64

void init_matrix(float* mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = 2.0f * ((float)rand() / RAND_MAX) - 1.0f;
    }
}

void elu_cpu(float* input, float* output, int m, int n, float alpha) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            int idx = i * n + j;
            output[idx] = input[idx] > 0.0f ? input[idx] : alpha * (expf(input[idx]) - 1.0f);
        }
    }
}

__global__ void elu_gpu(float* input, float* output, int m, int n, float alpha) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= m) return;
    for (int j = 0; j < n; j++) {
        int idx = row * n + j;
        output[idx] = input[idx] > 0.0f ? input[idx] : alpha * (expf(input[idx]) - 1.0f);
    }
}

int main() {
    float *h_A, *h_C_cpu, *h_C_gpu;
    float *d_A, *d_C;
    float alpha = 1.0f;

    int size_A = M * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    h_A = (float*)malloc(size_A);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    srand(time(NULL));
    init_matrix(h_A, M, N);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, 1);
    dim3 gridDim((M + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

    elu_gpu<<<gridDim, blockDim>>>(d_A, d_C, M, N, alpha);
    hipDeviceSynchronize();
    hipMemcpy(h_C_gpu, d_C, size_C, hipMemcpyDeviceToHost);

    elu_cpu(h_A, h_C_cpu, M, N, alpha);

    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_C_cpu[i] - h_C_gpu[i]) > 1e-4) {
            correct = false;
            break;
        }
    }
    
    printf("Results are %s\n", correct ? "correct" : "incorrect");

    free(h_A);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_C);

    return 0;
}
