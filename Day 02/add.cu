#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>

#define N 10000000
#define BLOCK_SIZE 256

void init_vector(float* vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

void vector_add_cpu(float* a, float* b, float* c, int n) {
    for (int i = 0; i < n; i++) {
        c[i] = a[i] + b[i];
    }
}

__global__ void vector_add_gpu(float* a, float* b, float* c, int n) {
    int block_offset = blockIdx.x * blockDim.x;
    int thread_offset = threadIdx.x;
    int index = block_offset + thread_offset;
    // index is the global thread index
    if (index < n) {
        c[index] = a[index]+b[index];
    }
}

int main() {
    float *h_a, *h_b, *h_c_cpu, *h_c_gpu;
    float *d_a, *d_b, *d_c;

    size_t size = N * sizeof(float);

    // Allocate memory for arrays on the CPU
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c_cpu = (float*)malloc(size);
    h_c_gpu = (float*)malloc(size);

    srand(time(NULL));

    init_vector(h_a, N);
    init_vector(h_b, N);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    vector_add_gpu<<<num_blocks, BLOCK_SIZE>>>(d_a, d_b, d_c, N);
    hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);
    

    // Validate results
    vector_add_cpu(h_a, h_b, h_c_cpu, N);

    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }

    printf("Results are %s\n", correct ? "correct" : "incorrect");


    // Free resources
    free(h_a);
    free(h_b);
    free(h_c_cpu);
    free(h_c_gpu);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}