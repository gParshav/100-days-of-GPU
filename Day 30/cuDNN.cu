#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define N 200000
#define BLOCK_SIZE 256

void init_vector(float* vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX * 2.0f - 1.0f; // Range [-1, 1]
    }
}

void relu_cpu(float *input, float *output, size_t length) {
    for (size_t i = 0; i < length; i++) {
        output[i] = (input[i] > 0) ? input[i] : 0;
    }
}

int main() {
    float *h_in, *h_out_cpu, *h_out_gpu;
    float *d_in, *d_out;

    size_t size = N * sizeof(float);
    
    h_in = (float*)malloc(size);
    h_out_cpu = (float*)malloc(size);
    h_out_gpu = (float*)malloc(size);

    srand(time(NULL));
    init_vector(h_in, N);

    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnTensorDescriptor_t tensorDesc;
    hipdnnCreateTensorDescriptor(&tensorDesc);
    hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, N);

    hipdnnActivationDescriptor_t actDesc;
    hipdnnCreateActivationDescriptor(&actDesc);
    hipdnnSetActivationDescriptor(actDesc, HIPDNN_ACTIVATION_RELU, HIPDNN_PROPAGATE_NAN, 0);

    float alpha = 1.0f, beta = 0.0f;
    hipdnnActivationForward(cudnn, actDesc, &alpha, tensorDesc, d_in, &beta, tensorDesc, d_out);

    hipMemcpy(h_out_gpu, d_out, size, hipMemcpyDeviceToHost);

    relu_cpu(h_in, h_out_cpu, N);

    bool match = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_out_cpu[i] - h_out_gpu[i]) > 1e-5) { 
            match = false;
            break;
        }
    }

    if (match) {
        printf("CPU and GPU outputs match.\n");
    } else {
        printf("CPU and GPU outputs do not match.\n");
    }

    hipdnnDestroyActivationDescriptor(actDesc);
    hipdnnDestroyTensorDescriptor(tensorDesc);
    hipdnnDestroy(cudnn);
    
    free(h_in);
    free(h_out_cpu);
    free(h_out_gpu);
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}