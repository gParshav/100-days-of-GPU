#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipDNN.h>

#define N 10000
#define DROPOUT_PROB 0.5f  

void init_vector(float* vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX; 
    }
}

int main() {
    float *h_in, *h_out_gpu;
    float *d_in, *d_out, *d_states;
    size_t size = N * sizeof(float);

    h_in = (float*)malloc(size);
    h_out_gpu = (float*)malloc(size);
    srand(time(NULL));
    init_vector(h_in, N);

    hipMalloc(&d_in, size);
    hipMalloc(&d_out, size);
    hipMemcpy(d_in, h_in, size, hipMemcpyHostToDevice);

    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    hipdnnDropoutDescriptor_t dropoutDesc;
    hipdnnCreateDropoutDescriptor(&dropoutDesc);

    size_t stateSize;
    hipdnnDropoutGetStatesSize(cudnn, &stateSize);
    hipMalloc(&d_states, stateSize);

    hipdnnSetDropoutDescriptor(dropoutDesc, cudnn, DROPOUT_PROB, d_states, stateSize, 0);

    hipdnnTensorDescriptor_t tensorDesc;
    hipdnnCreateTensorDescriptor(&tensorDesc);
    hipdnnSetTensor4dDescriptor(tensorDesc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, 1, 1, 1, N);

    cudnnDropoutForward(cudnn, dropoutDesc, tensorDesc, d_in, tensorDesc, d_out, NULL, 0);

    hipMemcpy(h_out_gpu, d_out, size, hipMemcpyDeviceToHost);

    hipdnnDestroyDropoutDescriptor(dropoutDesc);
    hipdnnDestroyTensorDescriptor(tensorDesc);
    hipdnnDestroy(cudnn);

    free(h_in);
    free(h_out_gpu);
    hipFree(d_in);
    hipFree(d_out);
    hipFree(d_states);

    printf("Dropout applied successfully!\n");

    return 0;
}