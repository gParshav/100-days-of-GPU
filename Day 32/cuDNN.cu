#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <cudnn.h>

#define BATCH 2       // Batch size
#define CHANNELS 3    // Number of channels
#define HEIGHT 4      // Height of input
#define WIDTH 4       // Width of input

void init_data(float *data, int size) {
    for (int i = 0; i < size; i++) {
        data[i] = (float)rand() / RAND_MAX * 2.0f - 1.0f; // Random values between -1 and 1
    }
}

int main() {
    cudnnHandle_t cudnn;
    cudnnCreate(&cudnn);

    int size = BATCH * CHANNELS * HEIGHT * WIDTH;
    
    // Allocate host memory
    float *h_input = (float*)malloc(size * sizeof(float));
    float *h_output = (float*)malloc(size * sizeof(float));
    init_data(h_input, size);

    // Allocate device memory
    float *d_input, *d_output, *d_mean, *d_variance, *d_gamma, *d_beta;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output, size * sizeof(float));
    hipMalloc(&d_mean, CHANNELS * sizeof(float));
    hipMalloc(&d_variance, CHANNELS * sizeof(float));
    hipMalloc(&d_gamma, CHANNELS * sizeof(float));
    hipMalloc(&d_beta, CHANNELS * sizeof(float));

    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);

    // Create tensor descriptor
    cudnnTensorDescriptor_t inputDesc;
    cudnnCreateTensorDescriptor(&inputDesc);
    cudnnSetTensor4dDescriptor(inputDesc, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, BATCH, CHANNELS, HEIGHT, WIDTH);

    // Perform Batch Normalization
    float alpha = 1.0f, beta = 0.0f;
    cudnnBatchNormalizationForwardInference(
        cudnn, CUDNN_BATCHNORM_SPATIAL, &alpha, &beta,
        inputDesc, d_input, inputDesc, d_output,
        inputDesc, d_gamma, d_beta, d_mean, d_variance, 1e-5);

    // Copy output back
    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    cudnnDestroyTensorDescriptor(inputDesc);
    cudnnDestroy(cudnn);
    free(h_input);
    free(h_output);
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_mean);
    hipFree(d_variance);
    hipFree(d_gamma);
    hipFree(d_beta);

    return 0;
}