#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include<hip/hip_runtime.h>

#define N 10000000
#define BLOCK_SIZE 256

void init_vector(float* vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = (float)rand() / RAND_MAX;
    }
}

void mish_cpu(float* x, float* y, int n) {
    for (int i = 0; i < n; i++) {
        float exp_x = exp(x[i]);          
        float softplus = log(1.0f + exp_x); 
        y[i] = x[i] * tanh(softplus);    
    }
}

__global__ void mish_gpu(float* x, float* y, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        float exp_x = expf(x[idx]);
        float softplus = logf(1.0f + exp_x);
        y[idx] = x[idx] * tanhf(softplus);
    }
}

int main() {
    float *h_a, *h_c_cpu, *h_c_gpu;
    float *d_a, *d_c;

    size_t size = N * sizeof(float);

    // Allocate memory for arrays on the CPU
    h_a = (float*)malloc(size);
    h_c_cpu = (float*)malloc(size);
    h_c_gpu = (float*)malloc(size);

    srand(time(NULL));

    init_vector(h_a, N);

    hipMalloc(&d_a, size);
    hipMalloc(&d_c, size);
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);

    int num_blocks = (N + BLOCK_SIZE - 1) / BLOCK_SIZE;
    mish_gpu<<<num_blocks, BLOCK_SIZE>>>(d_a, d_c, N);
    hipMemcpy(h_c_gpu, d_c, size, hipMemcpyDeviceToHost);
    

    // Validate results
    mish_cpu(h_a, h_c_cpu, N);

    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (fabs(h_c_cpu[i] - h_c_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }

    printf("Results are %s\n", correct ? "correct" : "incorrect");


    // Free resources
    free(h_a);
    free(h_c_cpu);
    free(h_c_gpu);
    hipFree(d_a);
    hipFree(d_c);

    return 0;
}