#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define M 256
#define N 128
#define BLOCK_SIZE 64

// Initialize vector with random values
void init_vector(float* vec, int n) {
    for (int i = 0; i < n; i++) {
        vec[i] = -10.0f + (20.0f * rand()) / RAND_MAX;
    }
}

void init_matrix(float* mat, int rows, int cols){
    for(int i=0;i<rows*cols;i++){
        mat[i] = (float)rand() / RAND_MAX;
    }
}


// CPU implementation of softmax
void softmax_cpu(float* input, float* output, int m, int n) {
    
    for(int i=0;i<m;i++){
        float sum = 0.0f;
        float max_val = 0.0f;
        for (int j = 0; j < n; j++) {
            max_val = max(max_val, input[i*n+j]);
        }

        for (int j = 0; j < n; j++) {
            sum += expf(input[i*n+j] - max_val);
        }

        for (int j = 0; j < n; j++) {
            output[i*n+j] = expf(input[i*n+j] - max_val) / sum;
        }
    }
    
}

// Naïve CUDA softmax kernel
__global__ void softmax_gpu(float* input, float* output, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= m) return;

    float sum = 0.0f;
    float max_val = 0.0f;
    for (int j = 0; j < n; j++) {
        max_val = max(max_val, input[row*n+j]);
    }

    for (int j = 0; j < n; j++) {
        sum += expf(input[row*n+j] - max_val);
    }

    for (int j = 0; j < n; j++) {
        output[row*n+j] = expf(input[row*n+j] - max_val) / sum;
    }
}

__global__ void online_softmax_gpu(float* input, float* output, int m, int n) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= m) return;

    float sum = 0.0f;
    float max_val = 0.0f;
    for (int j = 0; j < n; j++) {
        float curr = input[row*n+j];
        if(curr>max_val){
            sum = sum*expf(max_val - curr);
            max_val = curr;
        }

        sum+=expf(curr - max_val);
    }

    for (int j = 0; j < n; j++) {
        output[row*n+j] = expf(input[row*n+j] - max_val) / sum;
    }
}

int main() {

    float *h_A, *h_C_cpu, *h_C_gpu; //These will be stored on the CPU
    float *d_A, *d_C; //These will be stored on the GPU

    int size_A = M*N*sizeof(float);
    int size_C = M*N*sizeof(float);

    h_A = (float*)malloc(size_A);
    h_C_cpu = (float*)malloc(size_C);
    h_C_gpu = (float*)malloc(size_C);

    srand(time(NULL));
    init_matrix(h_A, M, N);

    hipMalloc(&d_A, size_A);
    hipMalloc(&d_C, size_C);

    hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, 1); //These are number of threads in x and y inside the block
    dim3 gridDim((M+BLOCK_SIZE-1)/BLOCK_SIZE, 1);


    online_softmax_gpu<<<gridDim, blockDim>>>(d_A, d_C, M, N);
    hipDeviceSynchronize();
    hipMemcpy(h_C_gpu, d_C, size_C, hipMemcpyDeviceToHost);

    // Compute softmax on CPU for validation
    softmax_cpu(h_A, h_C_cpu, M, N);

    // Validate results
    bool correct = true;
    for (int i = 0; i < N; i++) {
        
        if (fabs(h_C_cpu[i] - h_C_gpu[i]) > 1e-4) {
            correct = false;
            break;
        }
    }
    
    printf("Results are %s\n", correct ? "correct" : "incorrect");

    // Free memory
    free(h_A);
    free(h_C_cpu);
    free(h_C_gpu);
    hipFree(d_A);
    hipFree(d_C);

    return 0;

}