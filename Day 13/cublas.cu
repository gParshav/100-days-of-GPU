#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define m 128
#define k 128
#define n 128


void init_matrix(float* mat, int rows, int cols){
    for(int i=0;i<rows*cols;i++){
        mat[i] = (float)rand() / RAND_MAX;
    }
}

int main() {

    // Host matrices
    float *h_A = (float*)malloc(m * k * sizeof(float));
    float *h_B = (float*)malloc(k * n * sizeof(float));
    float *h_C = (float*)malloc(m * n * sizeof(float));

    init_matrix(h_A, m, k);
    init_matrix(h_B, k, n);

    // Device matrices
    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m * k * sizeof(float));
    hipMalloc(&d_B, k * n * sizeof(float));
    hipMalloc(&d_C, m * n * sizeof(float));

    // Create cuBLAS handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Copy matrices to device
    hipMemcpy(d_A, h_A, m * k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, k * n * sizeof(float), hipMemcpyHostToDevice);

    // Scaling factors
    float alpha = 1.0f;
    float beta = 0.0f;

    // Perform matrix multiplication: C = alpha*(A*B) + beta*C
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, m, k, &alpha, d_B, n, d_A, k, &beta, d_C, n);

    // Copy result back to host
    hipMemcpy(h_C, d_C, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // Cleanup
    hipblasDestroy(handle);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}